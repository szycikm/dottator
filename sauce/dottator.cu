#include <opencv2/opencv.hpp>
#include <sys/time.h>
#include "defines.h"
#include "utils.h"
#include "cvConvert.h"
#include "devFunctions.hpp"

int main(int argc, char *argv[])
{
	uint frameWidth, imgW, imgH, pixelCnt, framesW, framesH, blocksW, blocksH;
	float dotScaleFactor;
	char* inputFilename;
	char* outputFilename;
	char suffix[] = "_out.png";
	cv::Mat cvInImg;
	cv::Mat* cvOutImg;
	pixel_t* hostInPixels;
	pixel_t* devInPixels;
	uchar* hostOutPixels;
	uchar* devOutPixels;
	dim3* blocksPerGrid;
	dim3* threadsPerBlock;
	hipError_t err;

#ifdef DEBUG
	long startTime, endTime, startTimeKernel, endTimeKernel;
	struct timeval timecheck;
	gettimeofday(&timecheck, NULL);
	startTime = (long)timecheck.tv_sec * 1000000LL + (long)timecheck.tv_usec;
#endif

	if (argc < 3)
	{
		printf("Too few arguments\narg1: input filename\narg2: frame width (px)\narg3: dot scaling factor (default=1.0)\n");
		return 1;
	}

	// parse params
	inputFilename = argv[1];

	if(!fileExists(inputFilename))
	{
		printf("File doesn't exist\n");
		return 2;
	}

	outputFilename = (char*)malloc(strlen(inputFilename) + strlen(suffix) + 1);
	if (outputFilename == NULL)
	{
		printf("Can't allocate memory\n");
		goto unroll_outputFilename;
	}

	strcpy(outputFilename, inputFilename);
	strcat(outputFilename, suffix);

	frameWidth = atoi(argv[2]);

	dotScaleFactor = 1.0;
	if(argc >= 4)
		dotScaleFactor = atof(argv[3]);

	debug_printf("Input file:\t\t%s\nOutput file:\t\t%s\nFrame width:\t\t%dpx\nDot scaling factor:\t%f\n",
		inputFilename, outputFilename, frameWidth, dotScaleFactor);

	// load opencv image and convert it to array of pixels
	cvInImg = cv::imread(inputFilename);
	imgW = cvInImg.cols;
	imgH = cvInImg.rows;
	pixelCnt = imgW * imgH;
	cvOutImg = new cv::Mat(imgH, imgW, CV_8U);

	hostInPixels = (pixel_t*)malloc(pixelCnt * sizeof(pixel_t));
	if (hostInPixels == NULL)
	{
		printf("Can't allocate memory\n");
		goto unroll_hostInPixels;
	}

	hostOutPixels = (uchar*)malloc(pixelCnt * sizeof(pixel_t));
	if (hostOutPixels == NULL)
	{
		printf("Can't allocate memory\n");
		goto unroll_hostOutPixels;
	}

	cvToRawImg(&cvInImg, hostInPixels, imgH, imgW);

	// calculate number of frames and blocks
	framesW = imgW/frameWidth;
	if (imgW % frameWidth != 0) framesW++;

	framesH = imgH/frameWidth;
	if (imgH % frameWidth != 0) framesH++;

	blocksW = framesW/THREADS_DIM;
	if (framesW % THREADS_DIM != 0 || blocksW <= 0) blocksW++;

	blocksH = framesH/THREADS_DIM;
	if (framesH % THREADS_DIM != 0 || blocksH <= 0) blocksH++;

	debug_printf("imgW:\t\t\t%d\nimgH:\t\t\t%d\nframesW:\t\t%d\nframesH:\t\t%d\nblocksW:\t\t%d\nblocksH:\t\t%d\n",
		imgW, imgH, framesW, framesH, blocksW, blocksH);

	// copy memory to device
	if (hipMalloc((void**)&devInPixels, pixelCnt * sizeof(pixel_t)) != hipSuccess)
	{
		printf("Can't allocate GPU memory\n");
		goto unroll_devInPixels;
	}

	hipMemcpy(devInPixels, hostInPixels, pixelCnt * sizeof(pixel_t), hipMemcpyHostToDevice);

	if (hipMalloc((void**)&devOutPixels, pixelCnt * sizeof(uchar)) != hipSuccess)
	{
		printf("Can't allocate GPU memory\n");
		goto unroll_devOutPixels;
	}
	hipMemset(devOutPixels, 0, pixelCnt * sizeof(uchar)); // set all pixels to black

	// run kernel
	blocksPerGrid = new dim3(blocksW, blocksH);
	threadsPerBlock = new dim3(THREADS_DIM, THREADS_DIM);

#ifdef DEBUG
	gettimeofday(&timecheck, NULL);
	startTimeKernel = (long)timecheck.tv_sec * 1000000LL + (long)timecheck.tv_usec;
#endif

	dev_makeDots<<<*blocksPerGrid, *threadsPerBlock>>>(frameWidth, imgW, imgH, dotScaleFactor, devInPixels, devOutPixels);
	err = hipDeviceSynchronize();
	if (err != hipSuccess)
	{
		printf("Uh-oh, %s\n", hipGetErrorString(err));
		goto unroll_cudaerror;
	}

#ifdef DEBUG
	gettimeofday(&timecheck, NULL);
	endTimeKernel = (long)timecheck.tv_sec * 1000000LL + (long)timecheck.tv_usec;
	printf("Kernel execution took %ldus\n", endTimeKernel - startTimeKernel);
#endif

	// copy results from device
	hipMemcpy(hostOutPixels, devOutPixels, pixelCnt * sizeof(uchar), hipMemcpyDeviceToHost);

	rawToCvImg(hostOutPixels, cvOutImg, imgH, imgW);

	cv::imwrite(outputFilename, *cvOutImg);

unroll_cudaerror:
	delete threadsPerBlock;
	delete blocksPerGrid;
	hipFree(devOutPixels);
unroll_devOutPixels:
	hipFree(devInPixels);
unroll_devInPixels:
	free(hostOutPixels);
unroll_hostOutPixels:
	free(hostInPixels);
unroll_hostInPixels:
	delete cvOutImg;
	free(outputFilename);
unroll_outputFilename:

#ifdef DEBUG
	gettimeofday(&timecheck, NULL);
	endTime = (long)timecheck.tv_sec * 1000000LL + (long)timecheck.tv_usec;
	printf("Total execution took %ldus\n", endTime - startTime);
#endif

	return 0;
}
